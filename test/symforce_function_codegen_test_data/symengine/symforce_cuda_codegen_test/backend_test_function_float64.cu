
#include <hip/hip_runtime.h>
// -----------------------------------------------------------------------------
// This file was autogenerated by symforce from template:
//     function/FUNCTION.cu.jinja
// Do NOT modify by hand.
// -----------------------------------------------------------------------------

namespace sym {

__host__ __device__ void BackendTestFunctionFloat64(
    const double x, const double y, double* const __restrict__ res0,
    double* const __restrict__ res1, double* const __restrict__ res2,
    double* const __restrict__ res3, double* const __restrict__ res4,
    double* const __restrict__ res5, double* const __restrict__ res6,
    double* const __restrict__ res7, double* const __restrict__ res8,
    double* const __restrict__ res9, double* const __restrict__ res10,
    double* const __restrict__ res11, double* const __restrict__ res12,
    double* const __restrict__ res13, double* const __restrict__ res14,
    double* const __restrict__ res15, double* const __restrict__ res16,
    double* const __restrict__ res17, double* const __restrict__ res18,
    double* const __restrict__ res19, double* const __restrict__ res20,
    double* const __restrict__ res21, double* const __restrict__ res22,
    double* const __restrict__ res23, double* const __restrict__ res24,
    double* const __restrict__ res25, double* const __restrict__ res26,
    double* const __restrict__ res27, double* const __restrict__ res28,
    double* const __restrict__ res29, double* const __restrict__ res30,
    double* const __restrict__ res31, double* const __restrict__ res32,
    double* const __restrict__ res33, double* const __restrict__ res34,
    double* const __restrict__ res35, double* const __restrict__ res36,
    double* const __restrict__ res37, double* const __restrict__ res38,
    double* const __restrict__ res39, double* const __restrict__ res40,
    double* const __restrict__ res41, double* const __restrict__ res42,
    double* const __restrict__ res43, double* const __restrict__ res44,
    double* const __restrict__ res45, double* const __restrict__ res46,
    double* const __restrict__ res47, double* const __restrict__ res48,
    double* const __restrict__ res49, double* const __restrict__ res50,
    double* const __restrict__ res51, double* const __restrict__ res52,
    double* const __restrict__ res53, double* const __restrict__ res54,
    double* const __restrict__ res55, double* const __restrict__ res56,
    double* const __restrict__ res57, double* const __restrict__ res58,
    double* const __restrict__ res59) {
  // Total ops: 62

  // Intermediate terms (5)
  const double _tmp0 = M_LN2;
  const double _tmp1 = M_1_PI;
  const double _tmp2 = M_SQRT2;
  const double _tmp3 = sqrt(x);
  const double _tmp4 = x + y;

  // Output terms (60)
  if (res0 != nullptr) {
    *res0 = 0;
  }

  if (res1 != nullptr) {
    *res1 = 1;
  }

  if (res2 != nullptr) {
    *res2 = 1.0 / 2.0;
  }

  if (res3 != nullptr) {
    *res3 = 1.6000000000000001;
  }

  if (res4 != nullptr) {
    *res4 = 11.0 / 52.0;
  }

  if (res5 != nullptr) {
    *res5 = M_E;
  }

  if (res6 != nullptr) {
    *res6 = 1.0 / _tmp0;
  }

  if (res7 != nullptr) {
    *res7 = _tmp0;
  }

  if (res8 != nullptr) {
    *res8 = M_LN10;
  }

  if (res9 != nullptr) {
    *res9 = M_PI;
  }

  if (res10 != nullptr) {
    *res10 = M_PI_2;
  }

  if (res11 != nullptr) {
    *res11 = M_PI_4;
  }

  if (res12 != nullptr) {
    *res12 = _tmp1;
  }

  if (res13 != nullptr) {
    *res13 = 2 * _tmp1;
  }

  if (res14 != nullptr) {
    *res14 = M_2_SQRTPI;
  }

  if (res15 != nullptr) {
    *res15 = _tmp2;
  }

  if (res16 != nullptr) {
    *res16 = (1.0 / 2.0) * _tmp2;
  }

  if (res17 != nullptr) {
    *res17 = fabs(x);
  }

  if (res18 != nullptr) {
    *res18 = sin(x);
  }

  if (res19 != nullptr) {
    *res19 = cos(x);
  }

  if (res20 != nullptr) {
    *res20 = tan(x);
  }

  if (res21 != nullptr) {
    *res21 = asin(x);
  }

  if (res22 != nullptr) {
    *res22 = acos(x);
  }

  if (res23 != nullptr) {
    *res23 = atan(x);
  }

  if (res24 != nullptr) {
    *res24 = exp(x);
  }

  if (res25 != nullptr) {
    *res25 = log(x);
  }

  if (res26 != nullptr) {
    *res26 = sinh(x);
  }

  if (res27 != nullptr) {
    *res27 = cosh(x);
  }

  if (res28 != nullptr) {
    *res28 = tanh(x);
  }

  if (res29 != nullptr) {
    *res29 = floor(x);
  }

  if (res30 != nullptr) {
    *res30 = ceil(x);
  }

  if (res31 != nullptr) {
    *res31 = _tmp3;
  }

  if (res32 != nullptr) {
    *res32 = asinh(x);
  }

  if (res33 != nullptr) {
    *res33 = acosh(x);
  }

  if (res34 != nullptr) {
    *res34 = atanh(x);
  }

  if (res35 != nullptr) {
    *res35 = x - 5.5 * floor(0.18181818181818199 * x);
  }

  if (res36 != nullptr) {
    *res36 = x + 1;
  }

  if (res37 != nullptr) {
    *res37 = 2 * x;
  }

  if (res38 != nullptr) {
    *res38 = pow(x, 2);
  }

  if (res39 != nullptr) {
    *res39 = pow(x, 3);
  }

  if (res40 != nullptr) {
    *res40 = pow(x, 4);
  }

  if (res41 != nullptr) {
    *res41 = pow(x, 5);
  }

  if (res42 != nullptr) {
    *res42 = _tmp3;
  }

  if (res43 != nullptr) {
    *res43 = pow(x, 3.0 / 2.0);
  }

  if (res44 != nullptr) {
    *res44 = (1.0 / 2.0) * (((x) > 0) - ((x) < 0)) + 1.0 / 2.0;
  }

  if (res45 != nullptr) {
    *res45 = copysign(1.0, x);
  }

  if (res46 != nullptr) {
    *res46 = erfc(x);
  }

  if (res47 != nullptr) {
    *res47 = lgamma(x);
  }

  if (res48 != nullptr) {
    *res48 = erf(x);
  }

  if (res49 != nullptr) {
    *res49 = tgamma(x);
  }

  if (res50 != nullptr) {
    *res50 = atan2(x, y);
  }

  if (res51 != nullptr) {
    *res51 = fmax(x, y);
  }

  if (res52 != nullptr) {
    *res52 = fmin(x, y);
  }

  if (res53 != nullptr) {
    *res53 = x - y * floor(x / y);
  }

  if (res54 != nullptr) {
    *res54 = _tmp4;
  }

  if (res55 != nullptr) {
    *res55 = x * y;
  }

  if (res56 != nullptr) {
    *res56 = pow(x, y);
  }

  if (res57 != nullptr) {
    *res57 = pow(_tmp4, 2);
  }

  if (res58 != nullptr) {
    *res58 = pow(_tmp4, 3);
  }

  if (res59 != nullptr) {
    *res59 = copysign(x, y);
  }
}

}  // namespace sym
